/*
* Using thrust device_vectors is too heavy
* Resizes also call fill function to set values to 0
* reverting to using cub
* Required features.
*/
// #include "cuda_utils.h"
#include <iostream>
#include <string>
#include "../util/device_vector.h"
// #include <device_vector>
namespace cuslicer{

template<typename DATATYPE>
device_vector<DATATYPE>::device_vector(){
    allocated = 0;
    current_size = 0;
    free_size = 0;
}


template<typename DATATYPE>
device_vector<DATATYPE>::device_vector(std::vector<DATATYPE> &host){
    device_vector();
    resize(host.size());
    gpuErrchk(hipMemcpy(d->ptr(), host.data(), sizeof(DATATYPE) * current_size, hipMemcpyHostToDevice));
}

template<typename DATATYPE>
 void device_vector<DATATYPE>::resize(size_t new_size){
   if(new_size == 0)return clear();
   if(d ==nullptr){
     // First allocation
     d = cuda_memory::alloc(new_size);
     current_size = new_size;
     allocated = new_size;
     free_size = 0;
     return;
  }
  if(new_size > allocated){
    d = cuda_memory::alloc(new_size);
    allocated = new_size;
  }else{
    // std::cout <<"skipping allocation\n";
  }
  current_size = new_size;
  free_size = allocated-current_size;
}


template<typename DATATYPE>
 void device_vector<DATATYPE>::debug(std::string str){
    std::cout << str;
   if(d == nullptr){
     return;
   }
   hipSetDevice(0);
   DATATYPE * host = (DATATYPE *)malloc(sizeof(DATATYPE) * current_size);
   gpuErrchk(hipDeviceSynchronize());
   gpuErrchk(hipMemcpy(host, d->ptr(), sizeof(DATATYPE) * current_size, hipMemcpyDeviceToHost));
   for(int i = 0; i < current_size; i ++){
     std::cout << host[i] << " ";
   }
   std::cout << "\n";

   free(host);

}


template<typename DATATYPE>
device_vector<DATATYPE>::~device_vector(){
   // if(d != nullptr){
   //   gpuErrchk(hipFree(d));
   // }
}


template<typename DATATYPE>
device_vector<DATATYPE>& device_vector<DATATYPE>::operator=(device_vector<DATATYPE> &in){
   if(in.size()==0){
     this->resize(0);
     return *this;
   }
   this->resize(in.size());
   hipMemcpy(d->ptr(), in.ptr(), sizeof(DATATYPE) * in.size(), hipMemcpyDeviceToDevice);
   return *this;
}

template<typename DATATYPE>
void device_vector<DATATYPE>::append(device_vector<DATATYPE> &in){
    size_t start = this->size();
    this->resize(this->size() + in.size());
    hipMemcpy(&(d->ptr()[start]), in.ptr(), sizeof(DATATYPE) * in.size(), hipMemcpyDeviceToDevice);
}

template<typename DATATYPE>
bool device_vector<DATATYPE>::is_same(std::vector<DATATYPE> &expected){
  if(expected.size() != this->size())return false;
  DATATYPE * host = (DATATYPE *)malloc(sizeof(DATATYPE) * current_size);
  gpuErrchk(hipMemcpy(host, d->ptr(), sizeof(DATATYPE) * current_size, hipMemcpyDeviceToHost));
  bool is_correct = true;
  for(int i = 0; i < current_size; i ++){
    if(host[i] != expected[i]){
      is_correct = false;
      break;
    }
  }
  free(host);
  return is_correct;
}

template class device_vector <long>;

}
