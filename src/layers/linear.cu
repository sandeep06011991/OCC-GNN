#include "hip/hip_runtime.h"
#include "tensor.hh"
#include "linear.hh"
#include "nn_exception.hh"
#include <assert.h>
#include <iostream>

LinearLayer::LinearLayer(int dim1, int dim2){
  this->dim1 = dim1;
  this->dim2 = dim2;
  this->W = new Tensor<float>(allocate_random(dim1*dim2,dim2),dim1,dim2);
  this->b = new Tensor<float>(allocate_random(dim2,dim2),dim2,1);
  this->dW = new Tensor<float>(dim1,dim2);
  this->db = new Tensor<float>(dim2,1);

  this->in = *(new Tensor<float>(1,1));
}

LinearLayer::LinearLayer(float *W, float *B, int dim1, int dim2){
  this->dim1 = dim1;
  this->dim2 = dim2;
  this->W = new Tensor<float>(W,dim1,dim2);
  this->b = new Tensor<float>(B,dim2,1);
  this->dW = new Tensor<float>(dim1,dim2);
  this->db = new Tensor<float>(dim2,1);
  this->in = *(new Tensor<float>(1,1));
}

LinearLayer::LinearLayer(int dim1, int dim2, int in_dim){
    this->dim1 = dim1;
    this->dim2 = dim2;
    this->in_dim = in_dim;
    this->out = new Tensor<float>(in_dim,dim2);
    this->W = new Tensor<float>(allocate_random(dim1*dim2,dim2),dim1,dim2);
    this->b = new Tensor<float>(allocate_random(dim2,dim2),dim2,1);
    this->out_grad = new Tensor<float>(in_dim,dim1);
    this->dW = new Tensor<float>(allocate_random(dim1*dim2,dim2),dim1,dim2);
    this->db = new Tensor<float>(allocate_random(dim2,dim2),dim2,1);
    this->_btemp =  allocate_ones(in_dim,1);
    this->in = *(new Tensor<float>(1,1));

}


  void LinearLayer::update(float learning_rate){
    // std::cout << "Checking gradients \n";
    // this->dW->debugTensor();
    // this->dW->viewTensor();
    this->W->update(learning_rate, this->dW);
    this->b->update(learning_rate, this->db);
  }

LinearLayer::LinearLayer(float *W, float *B, int dim1, int dim2, int in_dim){
    this->dim1 = dim1;
    this->dim2 = dim2;
    this->in_dim = in_dim;
    this->out = new Tensor<float>(in_dim,dim2);
    this->W = new Tensor<float>(W,dim1,dim2);
    this->b = new Tensor<float>(B,dim2,1);
    this->out_grad = new Tensor<float>(in_dim,dim1);
    this->dW = new Tensor<float>(allocate_random(dim1*dim2,dim2),dim1,dim2);
    this->db = new Tensor<float>(allocate_random(dim2,dim2),dim2,1);
    this->_btemp =  allocate_ones(in_dim,1);
    this->in = *(new Tensor<float>(1,1));
}



// Copied code modifu this later to get grads as well
__global__ void cu_multiply(float* A, float* B, float * C,
                                    int rowsa, int colsa,
                                    int rowsb, int colsb,
                                    int rowsc, int colsc){
// Completely wrong. Redo this.
    __shared__ float sA[32][32];   // Tile size of 32x32
    __shared__ float sB[32][32];
    int Row = blockDim.x*blockIdx.x + threadIdx.x;
    int Col = blockDim.y*blockIdx.y + threadIdx.y;
    float Cvalue = 0.0;
    sA[threadIdx.x][threadIdx.y] = 0.0;
    sB[threadIdx.x][threadIdx.y] = 0.0;

    for (int k = 0; k < (((colsa - 1)/ 32) + 1); k++){
        if ( (Row < rowsa) && (threadIdx.y + (k*32)) < colsa){
            sA[threadIdx.x][threadIdx.y] = A[(Row*colsa) + threadIdx.y + (k*32)];
        }
        else{
            sA[threadIdx.x][threadIdx.y] = 0.0;
        }
        __syncthreads();
        if ( Col < colsb && (threadIdx.x + k*32) < rowsb){
          // Jumping . move access
            sB[threadIdx.x][threadIdx.y] = B[(threadIdx.x + k*32)*colsb + Col];
        }
        else{
            sB[threadIdx.x][threadIdx.y] = 0.0;
        }
        __syncthreads();

        for (int j = 0; j < 32; ++j){
            Cvalue += sA[threadIdx.x][j] * sB[j][threadIdx.y];
        }
        __syncthreads();
    }
    if (Row < rowsc && Col < colsc){
        C[Row*colsc + Col] = Cvalue;
    }
}

// CUDA PLAS
// a += b
// n is size of a
__global__ void cu_plus(float *A, const float *B, const int n){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while(tid < n){
		A[tid] = __fadd_rd(A[tid], B[tid]);
		tid += stride;
	}
}

__global__ void cu_add_bias(float *out, float *bias){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
	out[tid] = out[tid] + bias[threadIdx.x];
}

Tensor<float>& LinearLayer::forward(Tensor<float>& in_p){

  this->in_dim = in_p.dim1;
  this->in = in_p;
  Tensor<float>& in = in_p;
  int TILE_WIDTH = 32;

  // in.debugTensor();
  // W->debugTensor();
  if(this->out !=nullptr){
    delete this->out;
    delete this->out_grad;
    delete this->_btemp;
  }
  this->out = new Tensor<float>(in_dim,dim2);
  this->out_grad = new Tensor<float>(in_dim,dim1);
  this->_btemp =  allocate_ones(in_dim,1);

  mat_mul_a_b(in, true, *this->W, true , *this->out);
  NNException::throwIfDeviceErrorsOccurred("mat mul   failed");

  cu_add_bias<<<out->dim1, out->dim2>>>(out->data_device, b->data_device);
  NNException::throwIfDeviceErrorsOccurred("mat mul linear failed");

  hipDeviceSynchronize();
  return *out;
}

// void compute_w_grad(float *in, in_dim1, in_dim2, float *wx)

Tensor<float>& LinearLayer::backward(Tensor<float>& in_grad){
  assert(in_grad.dim1 = this->in_dim );
  assert(in_grad.dim2 = this->dim2);
  mat_mul_a_t_b(this->in,true, in_grad, true, *dW);
  // Tensor<float> * out_grad;
  mat_mul_a_b_t(in_grad,true,*W,true,*out_grad);
  mat_mul_a_t_b(in_grad,true,*this->_btemp,true,*db);
  hipDeviceSynchronize();
    // compute_w_grad(in_grad);
  return *out_grad;
}
