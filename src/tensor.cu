#include "tensor.hh"
#include "nn_exception.hh"
#include <random>
#include <assert.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

template<typename T>
Tensor<T>::Tensor( int dim1, int dim2){
  this->dim1 = dim1;
  this->dim2 = dim2;
  this->allocateMemory();
  hipDeviceSynchronize();
}

template<typename T>
Tensor<T>::Tensor(T* data, int dim1, int dim2){
  this->data_host = data;
  this->dim1 = dim1;
  this->dim2 = dim2;
  this->allocateMemory();
  this->copyHostToDevice();
  hipDeviceSynchronize();
}

template<typename T>
void Tensor<T>::allocateMemory(){
   hipMalloc(&this->data_device, dim1 * dim2 * sizeof(T));
   NNException::throwIfDeviceErrorsOccurred("memory allocation failed");
}

template<typename T>
void Tensor<T>::copyHostToDevice(){
   hipMemcpy(this->data_device,this->data_host,dim1 * dim2 * sizeof(T), hipMemcpyHostToDevice);
   NNException::throwIfDeviceErrorsOccurred("memory copy failed");
}

template<typename T>
void Tensor<T>::copyDeviceToHost(){
  if(this->data_host == nullptr){
     this->data_host = (T *)malloc(sizeof(T) * dim1 * dim2);
   }
   hipMemcpy(this->data_host,this->data_device,dim1 * dim2 * sizeof(T), hipMemcpyDeviceToHost);
   hipDeviceSynchronize();
   NNException::throwIfDeviceErrorsOccurred("memory copy failed");
}

template<typename T>
void Tensor<T>::debugTensor(){
  this->copyDeviceToHost();
  for(int i=0;i<4;i++){
    for(int j=0;j<4;j++){
      std::cout << this->data_host[i*this->dim2+j] << " ";
    }
    std::cout << "\n";
  }
  T s = 0;
  for(int i=0;i<this->dim1;i++){
    for(int j=0;j<this->dim2;j++){
      s += this->data_host[i*this->dim2+j] ;
    }
  }
  std::cout << "SUM " << s <<"\n";
}
float * allocate_random(int size){
  float * data = (float *)malloc(sizeof(float)*size);
  for(int i=0;i<size;i++){
    data[i] = rand();
  }
  return data;
}

// ,mat mul in row major format
void mat_mul_a_b(Tensor<float>& A, bool transA, Tensor<float>& B, bool transB,
                          Tensor<float>& C){
    float alpha = 1;
    float beta = 0;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasStatus_t success= hipblasSgemm( handle, HIPBLAS_OP_N , HIPBLAS_OP_N ,C.dim2, C.dim1
       , B.dim1 , \
       &alpha,   B.data_device, B.dim2, A.data_device, A.dim2 ,
        &beta , C.data_device, C.dim2);
    if ( success != HIPBLAS_STATUS_SUCCESS){
                std::cout << "\33[31mError: " << success << "\33[0m\n";
            hipblasDestroy(handle);
        }
}

// ,mat mul in row major format
void mat_mul_a_t_b(Tensor<float>& A, bool transA, Tensor<float>& B, bool transB,
                          Tensor<float>& C){
    float alpha = 1;
    float beta = 0;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasStatus_t success= hipblasSgemm( handle,  HIPBLAS_OP_N , HIPBLAS_OP_T  ,C.dim2, C.dim1
       , B.dim1  , \
       &alpha,   B.data_device, B.dim2, A.data_device, A.dim2 ,
        &beta , C.data_device, C.dim2);
    if ( success != HIPBLAS_STATUS_SUCCESS){
                std::cout << "\33[31mError: " << success << "\33[0m\n";
            hipblasDestroy(handle);
        }
}

// ,mat mul in row major format
void mat_mul_a_b_t(Tensor<float>& A, bool transA, Tensor<float>& B, bool transB,
                          Tensor<float>& C){
    float alpha = 1;
    float beta = 0;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasStatus_t success= hipblasSgemm( handle, HIPBLAS_OP_T , HIPBLAS_OP_N ,C.dim2, C.dim1
       , B.dim2 , \
       &alpha,   B.data_device, B.dim2, A.data_device, A.dim2 ,
        &beta , C.data_device, C.dim2);
    if ( success != HIPBLAS_STATUS_SUCCESS){
                std::cout << "\33[31mError: " << success << "\33[0m\n";
            hipblasDestroy(handle);
        }
}
