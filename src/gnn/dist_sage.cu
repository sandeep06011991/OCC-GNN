#include "hip/hip_runtime.h"
#include "util/dist_tensor.hh"
#include "gnn/dist_sage.hh"
#include <vector>
#include "util/tensor.hh"

void DistSageAggr::forward(vector<int>& ind_ptr, vector<int>& indices,
          DistTensor& in, int num_nodes_out, int num_nodes_in){
      // some magic reordering map
      int * reorder_map = (int *)malloc(sizeof(int) * num_nodes_out);
      for(int i=0; i< num_nodes_out;i++){
        reorder_map[i] = i%2;
      }

      if(this->out_feat == nullptr){
        this->out_feat->clearTensor();
        free(out_feat);
      }
      struct Shape s;
      s.dim1 = num_nodes_out;
      s.dim2 = in.s.dim2;
      out_feat = new DistTensor(s , reorder_map);

      // Populate remote_csrs.
      populateLocalGraphs(in, ind_ptr, indices);
      for(int i=0;i<no_gpus;i++){
        for(int j=0;j<no_gpus;j++){
        this->local_graph[i][j].create_csr();
        }
      }

      for(int i=0;i<no_gpus;i++){
        for(int j=0;j<no_gpus;j++){
          this->local_graph[i][j].forward(*(in.local_tensors[i]));
        }
      }

      // Create temporary tensors and clean up after wards.
      Tensor<float> * temp[4][4];
      for(int src=0;src<no_gpus;src++){
        for(int dest=0;dest<no_gpus;dest++){
          if(src!=dest) {
            temp[src][dest] = new Tensor<float>
                          (this->local_graph[src][dest]->out,
                            this->local_graph[src][dest]->dest_gpu);
          }else{
            temp[src][dest] = this->local_graph[src][dest]->out;
          }
        }
      }
      sync_all_gpu();
      for(int dest=0;dest<no_gpus;dest++){
        for(int src=0;src<no_gpus;src++){
          if(src!=dest) {
            merge(temp[dest][dest],temp[src][dest],this->local_graph[src][dest].local_to_local[id]);
          }
        }
      }


      out = new DistributedTensor(reorderer_map,shape);
      for(int i=0;i<4;i++){
        out.local_tensors[i] = temp[i][i];
      }
      for(int src=0;src<no_gpus;src++){
        for(int dest=0;dest<no_gpus;dest++){
          if(src!=dest) {
            temp[src][dest]->cleanUpTensor();
            free(temp[src][dest]);
          }
      }

}


void DistSageAggr::populateLocalGraphs(DistTensor &in, vector<int> &indptr,
                                          vector<int> &indices){
  for(int i=0;i<indptr.size()-1;i++){
    int nd1 = i;
    int dest_gpu = this->out_feat->global_to_gpu[nd1];
    int dest_local_id = this->out_feat->global_to_local[nd1];
    assert(dest_gpu < this->no_gpus);
    int start = indptr[i];
    int end = indptr[i+1];
    for(int j=start;j<end;j++){
      int nd2 = indices.at(j);
      int src_gpu = in.global_to_gpu[nd2];
      int src_local_id = in.global_to_local[nd2];
      this->local_graph[src_gpu][dest_gpu].add_edge(src_local_id,dest_local_id);
    }
  }
}

__global__ mergeKernel(float *src, int src_dim1, int src_dim2,
        float *dest, int dest_dim1, int dest_dim2,  int * indices, int size){
    int x = blockIdx.x;
    int y = threadIdx.x;
    dest[indices[x] * src_dim2 + y] += src[x * src_dim2 + y];
}


void merge(Tensor<float> *src, Tensor<float> *dest, Tensor<int> indices){
  assert(src->gpu_id == dest->gpu_id);
  assert(indices->gpu_id == dest->gpu_id);
  assert(src->gpu_id == dest->gpu_id);
  int noThreads = src.s.dim2;
  int noBlocks = src.s.dim1;
  hipSetDevice(src->gpu_id);
  <<<noBlocks,noThreads>>> mergeKernel(src->data, src.s.dim1, src.s.dim2,
                          dest->data, dest.s.dim1, dest.s.dim2,
                          indinces->data, indices.s.dim1);
}
