#include <iostream>
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>


using namespace std;

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

int main() {
    cuda_hello<<<1,1>>>();
    hipDeviceSynchronize();
    std::cout << "Hello world!\n";
    return 0;
}
