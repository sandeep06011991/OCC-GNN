#include "util/gpu.hh"
#include "nn_exception.hh"
#include <iostream>
void sync_all_gpus(){
  for(int i=0;i<no_gpus;i++){
      hipSetDevice(i);
      hipDeviceSynchronize();
  }
}


void enable_peer_communication(){
  for(int i=0;i<no_gpus;i++){
    hipSetDevice (i);
    for(int j=0;j<no_gpus;j++){
      if(i!=j){
        // std::cout << "cuda enable peer " << i << j <<"\n";
        int can_access_peer_0_1;
        hipDeviceCanAccessPeer(&can_access_peer_0_1, i, j);
        printf("hipDeviceCanAccessPeer(%d->%d): %d\n", i, j, can_access_peer_0_1);

        // hipDeviceEnablePeerAccess (j, i);
      }
    }
    NNException::throwIfDeviceErrorsOccurred("cuda peer comm setup failed\n");
  }
}
