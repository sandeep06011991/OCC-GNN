
#include <hip/hip_runtime.h>
#include<iostream>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


#include<iostream>

__global__
void f(int *a, size_t t){
	int id = threadIdx.x + (blockIdx.x * blockDim.x);
	while(id<t){
	a[id] = 0;
	id += (gridDim.x * blockDim.x);
	}
}
int main(){
	hipSetDevice(0);
	size_t size = 1 * 1024 * 1024 * 1024/sizeof(int);
	int *a, *b;
	//gpuErrchk(cudaMallocManaged(&a, size * sizeof(int)));
	gpuErrchk(hipHostAlloc(&b, size * sizeof(int), hipHostMallocMapped));
	gpuErrchk(hipHostGetDevicePointer(&a, b, 0));
	std::cout <<"Allocatoopn\n";
	f<<<1,1>>>(a,size);
	gpuErrchk(hipDeviceSynchronize());
	std::cout <<"kernel ok!\n";
	while(true){
	
	}
}
