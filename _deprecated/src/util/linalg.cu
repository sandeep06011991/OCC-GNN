#include "hipblas.h"

const char* cublasGetErrorString(hipblasStatus_t status)
{
    switch(status)
    {
        case HIPBLAS_STATUS_SUCCESS: return "HIPBLAS_STATUS_SUCCESS";
        case HIPBLAS_STATUS_NOT_INITIALIZED: return "HIPBLAS_STATUS_NOT_INITIALIZED";
        case HIPBLAS_STATUS_ALLOC_FAILED: return "HIPBLAS_STATUS_ALLOC_FAILED";
        case HIPBLAS_STATUS_INVALID_VALUE: return "HIPBLAS_STATUS_INVALID_VALUE";
        case HIPBLAS_STATUS_ARCH_MISMATCH: return "HIPBLAS_STATUS_ARCH_MISMATCH";
        case HIPBLAS_STATUS_MAPPING_ERROR: return "HIPBLAS_STATUS_MAPPING_ERROR";
        case HIPBLAS_STATUS_EXECUTION_FAILED: return "HIPBLAS_STATUS_EXECUTION_FAILED";
        case HIPBLAS_STATUS_INTERNAL_ERROR: return "HIPBLAS_STATUS_INTERNAL_ERROR";
    }
    return "unknown error";
}

// ,mat mul in row major format
void mat_mul_a_b(Tensor<float>& A, bool transA, Tensor<float>& B, bool transB,
                          Tensor<float>& C){
    float alpha = 1;
    float beta = 0;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasStatus_t success= hipblasSgemm( handle, HIPBLAS_OP_N , HIPBLAS_OP_N ,C.dim2, C.dim1
       , B.dim1 , \
       &alpha,   B.data_device, B.dim2, A.data_device, A.dim2 ,
        &beta , C.data_device, C.dim2);
    if ( success != HIPBLAS_STATUS_SUCCESS){
                std::cout << "\33[31mErrorjjj: " << cublasGetErrorString(success) << "\33[0m\n";

        }
      hipblasDestroy(handle);
}

// ,mat mul in row major format
void mat_mul_a_t_b(Tensor<float>& A, bool transA, Tensor<float>& B, bool transB,
                          Tensor<float>& C){
    float alpha = 1;
    float beta = 0;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasStatus_t success= hipblasSgemm( handle,  HIPBLAS_OP_N , HIPBLAS_OP_T  ,C.dim2, C.dim1
       , B.dim1  , \
       &alpha,   B.data_device, B.dim2, A.data_device, A.dim2 ,
        &beta , C.data_device, C.dim2);
    if ( success != HIPBLAS_STATUS_SUCCESS){
                std::cout << "\33[31mError: " << success << "\33[0m\n";

        }
    hipblasDestroy(handle);
}

// ,mat mul in row major format
void mat_mul_a_b_t(Tensor<float>& A, bool transA, Tensor<float>& B, bool transB,
                          Tensor<float>& C){
    float alpha = 1;
    float beta = 0;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasStatus_t success= hipblasSgemm( handle, HIPBLAS_OP_T , HIPBLAS_OP_N ,C.dim2, C.dim1
       , B.dim2 , \
       &alpha,   B.data_device, B.dim2, A.data_device, A.dim2 ,
        &beta , C.data_device, C.dim2);
    if ( success != HIPBLAS_STATUS_SUCCESS){
                std::cout << "\33[31mError: " << success << "\33[0m\n";

        }
    hipblasDestroy(handle);
}
