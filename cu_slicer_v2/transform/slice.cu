#include "hip/hip_runtime.h"
#include "slice.h"
#include <cstring>
#include "nvtx3/nvToolsExt.h"
#include "../util/cub.h"
#include "../util/array_utils.h"
using namespace cuslicer;


template<int BLOCK_SIZE, int TILE_SIZE>
__global__
void calculate_cache_hit_mask(long * in_nodes, int * storage_map, size_t size, int * cache_hit_mask, int * cache_miss_mask){
  int tileId = blockIdx.x;
  int last_tile = ((size - 1) / TILE_SIZE + 1);
  while(tileId < last_tile){
  int start = threadIdx.x + (tileId * TILE_SIZE);
  int end = min(static_cast<int64_t>(threadIdx.x + (tileId + 1) * TILE_SIZE), size);
  while(start < end){
        int tid = start;

   	long nd = in_nodes[tid];
  if(storage_map[nd] == -1){
		cache_hit_mask[tid] = 0;
		cache_miss_mask[tid] = 1;
	}else{
		cache_hit_mask[tid] = 1;
		cache_miss_mask[tid] = 0;
	 }
	 start = start + BLOCK_SIZE;
 }
 tileId += gridDim.x;
  }
}

template<int BLOCK_SIZE, int TILE_SIZE>
__global__
void  fill_cache_nodes(long * in_nodes, int * storage_map, size_t size, int * cache_hit_mask, int * cache_miss_mask, \
			long * miss_from , long* miss_to, long * hit_from, long *hit_to){
        int tileId = blockIdx.x;
        int last_tile = ((size - 1) / TILE_SIZE + 1);
        while(tileId < last_tile){
        int start = threadIdx.x + (tileId * TILE_SIZE);
        int end = min(static_cast<int64_t>(threadIdx.x + (tileId + 1) * TILE_SIZE), size);
  while(start < end){
        int tid = start;
        long nd = in_nodes[tid];
        if(storage_map[nd] == -1){
                miss_from[cache_miss_mask[tid]-1] = nd;
		            miss_to[cache_miss_mask[tid]-1] = tid;
        }else{
                hit_from[cache_hit_mask[tid]-1] = storage_map[nd];
                hit_to[cache_hit_mask[tid]-1] = tid;
        }
        start = start + BLOCK_SIZE;;

  }
    tileId += gridDim.x;
  }

}

void Slice::reorder(PartitionedLayer &l){\

    //   // Handle remote destination nodes
     for(int to = 0; to < this->num_gpus; to ++){
       // l.bipartite[to]->reorder_local(dr);
       dr->clear();
       dr->order(l.bipartite[to]->out_nodes_local);


       for(int from = 0; from < this->num_gpus; from++){
	       if(from == to) continue;
         dr->replace(l.bipartite[to]->push_from_ids[from]);
       }
     }

    for(int pull_from = 0;pull_from < this->num_gpus; pull_from++){
      dr->clear();
      dr->order(l.bipartite[pull_from]->in_nodes);
      for(int pull_to = 0; pull_to < this->num_gpus; pull_to ++ ){
        if(pull_from == pull_to)continue;
        dr->replace(l.bipartite[pull_from]->pull_to_ids[pull_to]);
    //     int start = l.bipartite[pull_to]->pull_from_offsets[pull_from];
    //     int end = l.bipartite[pull_to]->pull_from_offsets[pull_from + 1];
    //     thrust::device_vector<long> &f = l.bipartite[pull_from]->pull_to_ids[pull_to];
    //     thrust::device_vector<long> &t = l.bipartite[pull_to]->pulled_in_nodes;
    //     assert((end-start) <= t.size());
    //     f.clear();
    //     f.insert(f.end(), t.begin() + start, t.begin() + end);
    //     dr->replace(f);
    //   }
    }
    // nvtxRangePop();
  }



  void Slice::fill_cache_hits_and_misses(PartitionedSample &ps, int gpuid, device_vector<long> &in_nodes){
  	cache_hit_mask.clear();
  	cache_miss_mask.clear();
  	cache_hit_mask.resize(in_nodes.size());
  	cache_miss_mask.resize(in_nodes.size());
    calculate_cache_hit_mask<BLOCK_SIZE, TILE_SIZE><<<GRID_SIZE(in_nodes.size()), BLOCK_SIZE >>>(in_nodes.ptr(),\
  		       storage_map[gpuid].ptr(),\
  			in_nodes.size(),\
  			cache_hit_mask.ptr(),\
  		  cache_miss_mask.ptr());
    gpuErrchk(hipDeviceSynchronize());

    cuslicer::transform::self_inclusive_scan_int(cache_hit_mask);

    cuslicer::transform::self_inclusive_scan_int(cache_miss_mask);

    //  thrust::inclusive_scan(cache_hit_mask.begin(), cache_hit_mask.end(), cache_hit_mask.begin());
  	//  thrust::inclusive_scan(cache_miss_mask.begin(), cache_miss_mask.end(), cache_miss_mask.begin());
  	 int misses = cache_miss_mask[in_nodes.size()-1];
  	 int hits = cache_hit_mask[in_nodes.size() - 1];

  	 ps.cache_miss_from[gpuid].resize(misses);
     ps.cache_hit_from[gpuid].resize(hits);
     ps.cache_miss_to[gpuid].resize(misses);
     ps.cache_hit_to[gpuid].resize(hits);
     assert(hits + misses == in_nodes.size());
  	 fill_cache_nodes<BLOCK_SIZE, TILE_SIZE><<<GRID_SIZE(in_nodes.size()), BLOCK_SIZE>>>(in_nodes.ptr(),\
   		       storage_map[gpuid].ptr(),\
   			in_nodes.size(),\
   			cache_hit_mask.ptr(),\
   		  cache_miss_mask.ptr(),\
        ps.cache_miss_from[gpuid].ptr(), ps.cache_miss_to[gpuid].ptr(),\
        ps.cache_hit_from[gpuid].ptr(), ps.cache_hit_to[gpuid].ptr());
      gpuErrchk(hipDeviceSynchronize());

  }

  void Slice::slice_sample(Sample &s, PartitionedSample &ps){
    // Get local partitioning Map 
    // Todo: 
    // 1. Partition last layer of sample nodes into local partition ids. 
    auto nodes = s.block[s.num_layers]->layer_nds;
    this->sample_workload_map.resize(nodes.size());
    cuslicer::index_in<long,int>(nodes, this->workload_map, this->sample_workload_map);
    // this->workload_map
    // Get partitioned layers.
    for(int i= 1; i< s.num_layers + 1;i++){
        bool last_layer = false;
        if (i == s.num_layers) last_layer = true;
    	  PartitionedLayer& l = ps.layers[i-1];
        this->slice_layer(s.block[i-1]->layer_nds, \
            (* s.block[i]), l, last_layer);
        this->reorder(l);
        //consistency check
        for(int j = 0; j< this->num_gpus; j++){
          for(int k = 0; k < this->num_gpus;k ++ ){
            auto to = l.bipartite[j]->to_offsets[k + 1]  - l.bipartite[j]->to_offsets[k];
            assert(to == l.bipartite[k]->push_from_ids[j].size());
          }
        }
      }
      std::cout << "All clear\n";
      #ifdef DEBUG
        gpuErrchk(hipDeviceSynchronize());
      #endif
      // Not can further optimize this.
       for(int i=0;i<this->num_gpus;i++){
           ps.cache_miss_from[i].clear();
           ps.cache_hit_from[i].clear();
           ps.cache_miss_to[i].clear();
           ps.cache_hit_to[i].clear();
           ps.last_layer_nodes[i].clear();
           auto in_nodes = ps.layers[s.num_layers- 1].bipartite[i]->in_nodes;
           if(in_nodes.size() > 0){
              fill_cache_hits_and_misses(ps, i, in_nodes);
          }
          ps.last_layer_nodes[i] = ps.layers[0].bipartite[i]->out_nodes_local;
      }

      std::cout << "All clear\n";
}
