#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <iostream>
int main(){
	thrust::host_vector<int> v1;
   hipSetDevice(0);
   for(int i=100; i >0 ;  i--){
   	v1.push_back(i);
   }
   std::cout << v1.size();
   thrust::device_vector<int> nodes = v1;
    hipEvent_t event1, event2;
	hipEventCreate(&event1);
	hipEventCreate(&event2);
    hipEventRecord(event1,0);
    thrust::sort(nodes.begin(), nodes.end());
    auto it = thrust::unique(nodes.begin(), nodes.end());
    nodes.erase(it, nodes.end());
    hipEventRecord(event2,0);
    hipEventSynchronize(event2);
    float time;
    hipEventElapsedTime(&time, event1,event2);
    nvtxRangePushA(__FUNCTION__ ":nvtxRangePushA");
nvtxRangePop(); 

   std::cout <<"end\n";
}
