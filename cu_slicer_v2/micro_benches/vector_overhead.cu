#include <thrust/device_vector.h>
#include <iostream>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int main(){
	hipDeviceProp_t deviceProp;
hipGetDeviceProperties(&deviceProp, 0); // 0-th device
std::cout <<"sms:"<< deviceProp.multiProcessorCount <<"\n";
std::cout << "mts:" << deviceProp.maxThreadsPerMultiProcessor <<"\n";
hipSetDevice(0);
	thrust::device_vector<long> data;
	float *t;
	hipEvent_t event1;
    	hipEvent_t event2;
	gpuErrchk(hipEventCreate(&event1));
	gpuErrchk(hipEventCreate(&event2));
	float milliseconds;
	std::cout << "Rand" << sizeof(hiprandState) <<"\n";
	return 0;
	for(int i=0;i < 10; i++){
		for(int j=1; j <100000; j = j * 10){
			//gpuErrchk(hipEventRecord(event1));
			data.resize(j);
			//gpuErrchk(hipEventRecord(event2));
			//gpuErrchk(hipEventSynchronize(event2));
			//hipEventElapsedTime(&milliseconds, event1,event2);
			std::cout << i <<":"<< j <<":" << milliseconds <<"\n";
			
		}
	}
	hipDeviceSynchronize();

}
