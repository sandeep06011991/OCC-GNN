
#include <hip/hip_runtime.h>
#include <iostream>
#include <cassert>
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

int main(){
	std::cout << "Checl" <<  sizeof(hiprandState) <<" " << sizeof(int) <<  "\n";
	int * indptr_h;
	int * indptr_d;
	int num_nodes = 1000 * 1000 * 1000;
//gpuErrchk(cudaMalloc(&indptr_d, (num_nodes) * sizeof(long)));
gpuErrchk(hipHostAlloc(&indptr_h,(num_nodes + 1) * sizeof(long), hipHostMallocMapped | hipHostMallocWriteCombined ));
    //file1.read((char *)indptr_h,(this->num_nodes + 1) * sizeof(NDTYPE));
    gpuErrchk(hipHostGetDevicePointer(&indptr_d, indptr_h, 0));
	while(true){
	} 
}
