#include "cub.h"


namespace cuslicer{
 
    template<typename T>
    cuslicer::device_vector<T> transform<T>::d_temp_storage;
    template<typename T>
    cuslicer::device_vector<T> transform<T>::d_temp_out;
    template<typename T>
    cuslicer::device_vector<T> transform<T>::temporary; 

template<typename T>
T transform<T>::reduce(cuslicer::device_vector<T> & data_d){
    assert(data_d.size() != 0);
    transform::d_temp_out.resize(1);

    int num_elements = data_d.size();
    // Determine temporary device storage requirements
    size_t temp_storage_bytes;
    gpuErrchk(hipcub::DeviceReduce::Sum(NULL, temp_storage_bytes, data_d.ptr(), transform<T>::d_temp_out.ptr(), num_elements));
    d_temp_storage.resize(temp_storage_bytes/(sizeof(T)) + 1);
    gpuErrchk(hipcub::DeviceReduce::Sum(d_temp_storage.ptr(), temp_storage_bytes, data_d.ptr(), transform<T>::d_temp_out.ptr(), num_elements));
    return d_temp_out[0];
}

template<typename T>
T transform<T>::reduce_d(T * data_d, int numel){
    assert(numel != 0);
    transform::d_temp_out.resize(1);

    int num_elements = numel;
    // Determine temporary device storage requirements
    size_t temp_storage_bytes;
    gpuErrchk(hipcub::DeviceReduce::Sum(NULL, temp_storage_bytes, data_d, transform<T>::d_temp_out.ptr(), num_elements));
    d_temp_storage.resize(temp_storage_bytes/(sizeof(T)) + 1);
    gpuErrchk(hipcub::DeviceReduce::Sum(d_temp_storage.ptr(), temp_storage_bytes, data_d, transform<T>::d_temp_out.ptr(), num_elements));
    return d_temp_out[0];
}


template<typename T>
void transform<T>::sort(cuslicer::device_vector<T> &in, cuslicer::device_vector<T> &out){
        assert(in.size() != 0  );
        int  num_items = in.size();          // e.g., 7
        T  *d_keys_in = in.ptr();         // e.g., [8, 6, 7, 5, 3, 0, 9]
        out.resize(num_items);
        T  *d_keys_out = out.ptr();        // e.g., [        ...        ]
        // Determine temporary device storage requirements
        size_t   temp_storage_bytes = 0;
        gpuErrchk(hipcub::DeviceRadixSort::SortKeys(NULL, temp_storage_bytes, d_keys_in, d_keys_out, num_items));

        transform<T>::d_temp_storage.resize(temp_storage_bytes/(sizeof(T)) + 1);
        // Run sorting operation
        gpuErrchk(hipcub::DeviceRadixSort::SortKeys(transform<T>::d_temp_storage.ptr(), temp_storage_bytes, d_keys_in, d_keys_out, num_items));
  }
template<typename T>
void transform<T>::unique(cuslicer::device_vector<T>& sorted_in, cuslicer::device_vector<T> & out){
    assert(sorted_in.size() != 0);
    // Declare, allocate, and initialize device-accessible pointers for input and output
    int  num_items = sorted_in.size();              // e.g., 8
    T  *d_in = sorted_in.ptr();                  // e.g., [0, 2, 2, 9, 5, 5, 5, 8]
    out.resize(num_items);
    T  *d_out = out.ptr();                 // e.g., [ ,  ,  ,  ,  ,  ,  ,  ]
    d_temp_out.resize(1);
    T  *d_num_selected_out = d_temp_out.ptr();    // e.g., [ ]

    // Determine temporary device storage requirements
    size_t   temp_storage_bytes = 0;
    gpuErrchk(hipcub::DeviceSelect::Unique(NULL, temp_storage_bytes, d_in, d_out, d_num_selected_out, num_items));
    transform<T>::d_temp_storage.resize(temp_storage_bytes/(sizeof(T)) + 1);
    // Run selection
    gpuErrchk(hipcub::DeviceSelect::Unique(transform<T>::d_temp_storage.ptr(), temp_storage_bytes, d_in, d_out, d_num_selected_out, num_items));
    out.resize(d_temp_out[0]);
    // d_out                 <-- [0, 2, 9, 5, 8]
    // d_num_selected_out    <-- [5]
  }
template<typename T>
void transform<T>::remove_duplicates(cuslicer::device_vector<T> &in, cuslicer::device_vector<T>& out){
      sort(in, temporary);
      unique(temporary, out);
  }

template<typename T>
void transform<T>::exclusive_scan(cuslicer::device_vector<T> &in, cuslicer::device_vector<T>& out){
    assert(in.size() != 0);
    int  num_items = in.size();      // e.g., 7
    T  *d_in = in.ptr();          // e.g., [8, 6, 7, 5, 3, 0, 9]
    out.resize(num_items);
    T  *d_out = out.ptr();         // e.g., [ ,  ,  ,  ,  ,  ,  ]
    // Determine temporary device storage requirements
    size_t   temp_storage_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(NULL, temp_storage_bytes, d_in, d_out, num_items);
    transform<T>::d_temp_storage.resize(temp_storage_bytes/(sizeof(T)) + 1);
    // Allocate temporary storage
    // Run exclusive prefix sum
    hipcub::DeviceScan::ExclusiveSum(transform<T>::d_temp_storage.ptr(), temp_storage_bytes, d_in, d_out, num_items);
  }

  template<typename T>
  void transform<T>::inclusive_scan(cuslicer::device_vector<T> &in, cuslicer::device_vector<T>& out){
      assert(in.size() != 0);
      int  num_items = in.size();      // e.g., 7
      T  *d_in = in.ptr();          // e.g., [8, 6, 7, 5, 3, 0, 9]
      out.resize(num_items);
      T  *d_out = out.ptr();         // e.g., [ ,  ,  ,  ,  ,  ,  ]
      // Determine temporary device storage requirements
      size_t   temp_storage_bytes = 0;
      hipcub::DeviceScan::InclusiveSum(NULL, temp_storage_bytes, d_in, d_out, num_items);
      transform<T>::d_temp_storage.resize(temp_storage_bytes/(sizeof(T)) + 1);
      // Allocate temporary storage
      // Run exclusive prefix sum
      hipcub::DeviceScan::InclusiveSum(transform<T>::d_temp_storage.ptr(), temp_storage_bytes, d_in, d_out, num_items);
    }

  template<typename T>
    void transform<T>::self_inclusive_scan(cuslicer::device_vector<T> &in){
        assert(in.size() != 0);
        int  num_items = in.size();      // e.g., 7
        T  *d_in = in.ptr();          // e.g., [8, 6, 7, 5, 3, 0, 9]
        T  *d_out = in.ptr();         // e.g., [ ,  ,  ,  ,  ,  ,  ]
        // Determine temporary device storage requirements
        size_t   temp_storage_bytes = 0;
        hipcub::DeviceScan::InclusiveSum(NULL, temp_storage_bytes, d_in, d_out, num_items);
        d_temp_storage.resize(temp_storage_bytes/(sizeof(T)) + 1);
        // Allocate temporary storage
        // Run exclusive prefix sum
        hipcub::DeviceScan::InclusiveSum(d_temp_storage.ptr(), temp_storage_bytes, d_in, d_out, num_items);
      }

      // // Use Templates
      // template<typename T>
      // void transform<T>::self_inclusive_scan_int(cuslicer::device_vector<int> &in){
      //     assert(in.size() != 0);
      //     int  num_items = in.size();      // e.g., 7
      //     int  *d_in = in.ptr();          // e.g., [8, 6, 7, 5, 3, 0, 9]
      //     int  *d_out = in.ptr();         // e.g., [ ,  ,  ,  ,  ,  ,  ]
      //     // Determine temporary device storage requirements
      //     size_t   temp_storage_bytes = 0;
      //     hipcub::DeviceScan::InclusiveSum(NULL, temp_storage_bytes, d_in, d_out, num_items);
      //     d_temp_storage.resize(temp_storage_bytes/(sizeof(int)) + 1);
      //     // Allocate temporary storage
      //     // Run exclusive prefix sum
      //     hipcub::DeviceScan::InclusiveSum(d_temp_storage.ptr(), temp_storage_bytes, d_in, d_out, num_items);
      //   }
 
  template class transform<NDTYPE>;
  // template class transform<PARTITIONIDX>;
}
