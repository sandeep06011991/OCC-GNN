
#include <hip/hip_runtime.h>
#include<chrono>
#include <iostream>
#include <string>
#include <cstring>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int main(){
 auto start1 = std::chrono::system_clock::now();
     for(int i=0;i<100;i++){
        for(int j=0;j<4;j++){
                hipSetDevice(j);
                // void * device_array;
                // int num_bytes = 1<<10;
                // gpuErrchk(cudaMalloc((void**)&device_array, num_bytes));
                // cudaFree(device_array);
             hipDeviceSynchronize();
            }
        }
  auto end1 = std::chrono::system_clock::now();
  std::chrono::duration<double> elapsed1 = end1 - start1;
  std::cout << "DeviceSync: " << elapsed1.count()/100 << "s\n";
  hipDeviceSynchronize();
  hipStream_t stream[4];
  for(int j=0;j<4;j++){
    (hipStreamCreate(&stream[j]));
  }
  auto start2 = std::chrono::system_clock::now();
     for(int j=0;j<4;j++){
       auto start2= std::chrono::system_clock::now();
       for(int i=0;i<100;i++){
         hipSetDevice(j);
         gpuErrchk(hipStreamSynchronize(stream[j]));
     }
   }
   auto end2 = std::chrono::system_clock::now();
   std::chrono::duration<double> elapsed2 = end2 - start2;
   std::cout << "Stream sync: " << elapsed2.count()/100 << "s\n";

   int num_bytes = 1024 * 1024 * 1024;
   hipStream_t stream1;
   hipStreamCreate(&stream1);
   void * device_array;
   void * host_array = (void *)malloc(num_bytes);
   gpuErrchk(hipMalloc((void**)&device_array, num_bytes));
   for(int i=1024;i<1024*1024*1024;i=i*8){
     auto start3= std::chrono::system_clock::now();
     gpuErrchk(hipMemcpyAsync(device_array,host_array,i,hipMemcpyHostToDevice,stream1));
     hipStreamSynchronize(stream1);
     auto end3 = std::chrono::system_clock::now();
     std::chrono::duration<double> elapsed3 = end3 - start3;
     std::cout << "data moevemt: "<< i * 1.0/(1024 * 1024) <<"MB "<< elapsed3.count() << "s\n";

   }
}
