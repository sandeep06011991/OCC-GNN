
#include <hip/hip_runtime.h>
#include<iostream>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
// Check that I can do a p2p copy
float p2p_copy (size_t size)
{
  //size = sizeof(int);
  hipSetDevice (0);
  for(size_t i=1;i< 1024 * 4;i = i * 4){
      hipEvent_t begin, end;
	std::cout << "size in MB" << i <<"\n";

      hipEventCreate (&begin);
      hipEventCreate (&end);
	void * host;
      //void *host = malloc(i * 1024 * 1024);
      hipHostMalloc(&host, i * 1024 * 1024, hipHostMallocDefault);
      void *device;
      gpuErrchk(hipMalloc(&device,i * 1024 * 1024));
      for(int j=0;j<4;j++){
      hipEventRecord (begin);
      hipMemcpy(device, host, i * 1024 * 1024,hipMemcpyHostToDevice);
      hipEventRecord (end);
      hipEventSynchronize (end);
      float elapsed;
      hipEventElapsedTime (&elapsed, begin, end);
      elapsed /= 1000;
      std::cout << "Data movement" << i << ": " <<  elapsed << " : " << (i * (1.0)/1024)/elapsed <<" GBps\n";
      }
      hipFree(device);
      hipFree(host);
      hipEventDestroy (end);
      hipEventDestroy (begin);
  }
}

int main(){
  p2p_copy(1024 * 1024 * 1024);
}
