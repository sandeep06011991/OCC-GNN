
#include <hip/hip_runtime.h>
#include<iostream>
// Check that I can do a p2p copy 
float p2p_copy (size_t size)
{
  int *pointers[2];

  size = sizeof(int);
  hipSetDevice (0);
  hipDeviceEnablePeerAccess (1, 0);
  hipMalloc (&pointers[0], size);
  int t = 1991;
  hipMemcpy(pointers[0], &t, size ,hipMemcpyHostToDevice);

  hipSetDevice (1);
  hipDeviceEnablePeerAccess (0, 1);
  hipMalloc (&pointers[1], size);

  hipEvent_t begin, end;
  hipEventCreate (&begin);
  hipEventCreate (&end);

  hipEventRecord (begin);
  hipMemcpy (pointers[1], pointers[0],size, hipMemcpyDeviceToDevice);
  hipEventRecord (end);
  int c;
  hipMemcpy(&c,pointers[1],sizeof(int), hipMemcpyDeviceToHost);
  hipEventSynchronize (end);

  float elapsed;
  hipEventElapsedTime (&elapsed, begin, end);
  elapsed /= 1000;

  hipSetDevice (0);
  hipFree (pointers[0]);

  hipSetDevice (1);
  hipFree (pointers[1]);

  std::cout << c <<"\n";
  hipEventDestroy (end);
  hipEventDestroy (begin);

  return elapsed;
}

int main(){
  p2p_copy(10);
}
