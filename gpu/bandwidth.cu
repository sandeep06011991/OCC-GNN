
#include <hip/hip_runtime.h>
#include<iostream>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
// Check that I can do a p2p copy
float p2p_copy (size_t size)
{
  //size = sizeof(int);
  hipSetDevice (0);
  for(int i=1024;i<1024*1024*1024;i = i * 16){
      hipEvent_t begin, end;
      hipEventCreate (&begin);
      hipEventCreate (&end);
      void *host = malloc(i);
      void *device;
      gpuErrchk(hipMalloc(&device,i));
      hipEventRecord (begin);
      hipMemcpy(device, host, i,hipMemcpyHostToDevice);
      hipEventRecord (end);
      hipEventSynchronize (end);
      float elapsed;
      hipEventElapsedTime (&elapsed, begin, end);
      elapsed /= 1000;
      hipFree(device);
      free(host);
      std::cout << "Data movement" << i << " " << elapsed <<"\n";
      hipEventDestroy (end);
      hipEventDestroy (begin);
  }
}

int main(){
  p2p_copy(1024 * 1024 * 1024);
}
