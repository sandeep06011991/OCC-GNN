
#include <hip/hip_runtime.h>
#include<time.h>
#include<iostream>
int main(){
	float *a, *b;
	size_t size = (1024 * 1024 * 1024);
	hipSetDevice(0);
	hipMalloc(&a, size);
	hipSetDevice(1);
	hipMalloc(&b, size);
	float *c;
	hipHostMalloc(&c, size, hipHostMallocDefault);
	for(int i=0;i<10;i++){
		clock_t begin_time = clock();
		hipSetDevice(0);
		hipMemcpy(a,b, size, hipMemcpyDeviceToDevice);
		hipDeviceSynchronize();
		float t =  float( clock () - begin_time ) /  CLOCKS_PER_SEC;
		std::cout << "Bandwidth" << 1/t << "GBps\n";
		begin_time = clock();
		hipMemcpy(a,c, size, hipMemcpyHostToDevice);
		hipDeviceSynchronize();
		t =  float( clock () - begin_time ) /  CLOCKS_PER_SEC;
                std::cout << "Host Bandwidth" << 1/t << "GBps\n";

	}

}
